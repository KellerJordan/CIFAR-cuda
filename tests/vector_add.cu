#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA Kernel
__global__ void vectorAdd(float *C, int N) {
    // Calculate global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float s = 0;
        for (int i = 0; i < N; i++) {
            if (C[i] > 0)
                s += 1;
        }
        C[idx] = 1 + s / N;
    }
}

int main() {
    // Size of vectors
    int N = 1 << 20; // 1 Million elements

    size_t size = N * sizeof(float);

    // Allocate host memory
    float *h_C = (float*)malloc(size);

    // Allocate device memory
    float *d_C;
    hipError_t err;

    err = hipMalloc((void**)&d_C, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Start timing using CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch the vectorAdd CUDA Kernel
    int threadsPerBlock = 1024;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_C, N);

    hipDeviceSynchronize();

    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time for 1000 vector additions: %f ms\n", milliseconds);
    
    // Check for any errors launching the kernel
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy device result vector C to host
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify the result
    printf("%f\n", h_C[1000]);
    float s = 0;
    for (int i = 0; i < N; i++) {
        s += h_C[i];
    }
    printf("%f\n", s / N);

    // Free device memory
    hipFree(d_C);

    // Free host memory
    free(h_C);

    // Reset the device and exit
    hipDeviceReset();

    bool success = true;
    return (success ? EXIT_SUCCESS : EXIT_FAILURE);
}
