
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<math.h>
#include<stdbool.h>

const int DIM = 3*32*32;
const int CLASSES = 10;
//const int N_TRAIN = 50000;
const int N_TRAIN = 500;
const int N_TEST = 10000;

__global__ void cuda_forward(float *x_ND, float *w_CD, float *o_NC) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N_TRAIN*CLASSES) {
        float sum = 0;
        int n = idx / CLASSES;
        int c = idx % CLASSES;
        for (int d = 0; d < DIM; d++) {
            sum += x_ND[n*DIM+d] * w_CD[c*DIM+d];
        }
        o_NC[idx] = sum;
    }
}

unsigned char* read_data(const char path[]) {

    FILE *file = fopen(path, "rb");
    if (file == NULL) {
        perror("Error opening file");
        exit(1);
    }

    fseek(file, 0, SEEK_END);
    long fileSize = ftell(file);
    rewind(file);

    // Allocate a buffer to hold the file contents
    unsigned char *buffer = (unsigned char *)malloc(fileSize);
    if (buffer == NULL) {
        perror("Memory error");
        fclose(file);
        exit(1);
    }

    // Read the file into the buffer
    size_t bytesRead = fread(buffer, 1, fileSize, file);
    if (bytesRead != fileSize) {
        perror("Error reading file");
    }

    // Clean up
    fclose(file);

    return buffer;
}

float hash(float *x) {
    float sum = 0;
    for (int i = 0; i < 1000; i+=2) {
        sum += abs(x[i]);
    }
    return sum;
}

float *forward_linear(float *x_ND, float *w_CD, float *o_NC, int num) {
    for (int n = 0; n < num; n++) {
        for (int c = 0; c < CLASSES; c++) {
            int idx = CLASSES * n + c;
            o_NC[idx] = 0;
            for (int d = 0; d < DIM; d++) {
                o_NC[idx] += x_ND[n*DIM+d] * w_CD[c*DIM+d];
            }
        }
    }
    return o_NC;
}

float *softmax(float *o_NC, int num) {
    float *z_NC = (float *)malloc(num*CLASSES*sizeof(float));
    for (int n = 0; n < num; n++) {
        float Z = 0;
        for (int c = 0; c < CLASSES; c++) {
            z_NC[n*CLASSES+c] = expf(o_NC[n*CLASSES+c]);
            Z += z_NC[n*CLASSES+c];
        }
        for (int c = 0; c < CLASSES; c++) {
            z_NC[n*CLASSES+c] /= Z;
        }
    }
    return z_NC;
}

float *one_hot(long *y, int num) {
    float *z = (float *)malloc(num*CLASSES*sizeof(float));
    for (int n = 0; n < num; n++) {
        for (int c = 0; c < CLASSES; c++) {
            if (y[n] == c) {
                z[n*CLASSES+c] = 1;
            } else {
                z[n*CLASSES+c] = 0;
            }
        }
    }
    return z;
}

float *sub(float *x1, float *x2, int num) {
    for (int i = 0; i < num; i++) {
        x1[i] -= x2[i];
    }
    return x1;
}

float cross_entropy(float *p_NC, long *y_N, int num) {
    float loss = 0;
    for (int n = 0; n < num; n++) {
        if (!(y_N[n] >= 0 && y_N[n] < 10)) {
            printf("y[n] out of range: %ld\n", y_N[n]);
            exit(1);
        }
        loss += -logf(p_NC[n*CLASSES+y_N[n]]);
    }
    return loss;
}

const float ETA = 0.01/N_TRAIN;
float *fit_linear(float *x_ND, long *y_N) {

    hipError_t err;
    size_t size;

    struct timespec start, end;
    double elapsed;

    float *xT_DN = (float *)malloc(N_TRAIN*DIM*sizeof(float));
    for (int n = 0; n < N_TRAIN; n++) {
        for (int d = 0; d < DIM; d++) {
            xT_DN[d*N_TRAIN+n] = x_ND[n*DIM+d];
        }
    }

    float *w_CD = (float *)malloc(CLASSES*DIM*sizeof(float));
    for (int c = 0; c < CLASSES; c++)
        for (int d = 0; d < DIM; d++)
            w_CD[c*DIM+d] = 0;

    float *xc_ND;
    size = N_TRAIN*DIM*sizeof(float);
    err = hipMalloc((void**)&xc_ND, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(xc_ND, x_ND, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector C from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *wc_CD;
    size = CLASSES*DIM*sizeof(float);
    err = hipMalloc((void**)&wc_CD, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(wc_CD, w_CD, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector C from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *oc_NC;
    size = N_TRAIN*CLASSES*sizeof(float);
    err = hipMalloc((void**)&oc_NC, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    float *o_NC = (float *)malloc(size);

    int steps = 200;
    for (int step = 0; step < steps; step++) {

        clock_gettime(CLOCK_MONOTONIC, &start);

        int N = N_TRAIN*CLASSES;
        int threadsPerBlock = 1024;
        int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
        cuda_forward<<<blocksPerGrid, threadsPerBlock>>>(xc_ND, wc_CD, oc_NC);
        err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        size = N_TRAIN*CLASSES*sizeof(float);
        err = hipMemcpy(o_NC, oc_NC, size, hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        float *p_NC = softmax(o_NC, N_TRAIN);
        float *delta_NC = sub(one_hot(y_N, N_TRAIN), p_NC, N_TRAIN*CLASSES);
        float loss = cross_entropy(p_NC, y_N, N_TRAIN);

        float *u_CD = (float *)malloc(CLASSES*DIM*sizeof(float));
        for (int c = 0; c < CLASSES; c++)
            for (int d = 0; d < DIM; d++)
                u_CD[c*DIM+d] = 0;

        for (int c = 0; c < CLASSES; c++) {
            for (int d = 0; d < DIM; d++) {
                for (int n = 0; n < N_TRAIN; n++) {
                    u_CD[c*DIM+d] += delta_NC[n*CLASSES+c] * xT_DN[d*N_TRAIN+n];
                    //u_CD[c*DIM+d] += delta_NC[n*CLASSES+c] * x_ND[n*DIM+d]; // 6x slower bc of bad memory access pattern
                }
            }
        }
        for (int c = 0; c < CLASSES; c++)
            for (int d = 0; d < DIM; d++)
                w_CD[c*DIM+d] += ETA * u_CD[c*DIM+d];

        size = CLASSES*DIM*sizeof(float);
        err = hipMemcpy(wc_CD, w_CD, size, hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to copy vector C from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        free(u_CD);

        free(p_NC);
        free(delta_NC);

        clock_gettime(CLOCK_MONOTONIC, &end);
        elapsed = (end.tv_sec - start.tv_sec);
        elapsed += (end.tv_nsec - start.tv_nsec) / 1e9;
        printf("Time elapsed: %.6f seconds\n", elapsed);

        printf("Step: %d, Loss: %f\n", step, loss/N_TRAIN);
    }


    free(xT_DN);
    free(o_NC);
    hipFree(xc_ND);
    hipFree(wc_CD);
    hipFree(oc_NC);

    return w_CD;
}

int eval_linear(float *w_CD, float *x_MD, long *y_M) {
    float *o_MC = (float *)malloc(N_TEST*CLASSES*sizeof(float));
    forward_linear(x_MD, w_CD, o_MC, N_TEST);
    int correct = 0;
    for (int m = 0; m < N_TEST; m++) {
        int max_i = 0;
        int max_v = -100000;
        for (int c = 0; c < CLASSES; c++) {
            if (o_MC[m*CLASSES+c] > max_v) {
                max_i = c;
                max_v = o_MC[m*CLASSES+c];
            }
        }
        if (max_i == y_M[m]) {
            correct += 1;
        }
    }
    free(o_MC);
    return correct;
}

int main() {
    float *train_x_ND = (float *)read_data("/home/ubuntu/notebooks/train_x.bin");
    long *train_y_N = (long *)read_data("/home/ubuntu/notebooks/train_y.bin");
    float *weight_CD = fit_linear(train_x_ND, train_y_N);
    free(train_x_ND);
    free(train_y_N);

    float *test_x_MD = (float *)read_data("/home/ubuntu/notebooks/test_x.bin");
    long *test_y_M = (long *)read_data("/home/ubuntu/notebooks/test_y.bin");

    int correct = eval_linear(weight_CD, test_x_MD, test_y_M);
    printf("Correct: %d\n", correct);

    free(test_x_MD);
    free(test_y_M);
    free(weight_CD);

    hipDeviceReset();

    return 0;
}
