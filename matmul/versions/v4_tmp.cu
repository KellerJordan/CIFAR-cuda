// 3031 Gflops

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<math.h>
#include<stdbool.h>

float *cpu_matrix(int n) {
    size_t size = n * n * sizeof(float);
    float *M = (float *)malloc(size);
    return M;
}

void zero_init(float *M, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            M[n*i+j] = 0;
        }
    }
}

void rand_init(float *M, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            float random_float = (float)rand() / (float)RAND_MAX;
            M[n*i+j] = random_float;
        }
    }
}

float *cuda_matrix(int n) {
    size_t size = n * n * sizeof(float);
    float *dM;
    hipError_t err = hipMalloc((void**)&dM, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    return dM;
}

void cpu_to_cuda(float *hM, float *dM, int n) {
    size_t size = n*n*sizeof(float);
    hipError_t err = hipMemcpy(dM, hM, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "line 23 Failed to copy vector C from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

const int BLOCKSIZE = 32;

__global__ void cuda_matmul(float *A, float *B, float *C, int n) {
    int i0 = blockIdx.x * BLOCKSIZE + 2 * threadIdx.x / 32;
    int j0 = blockIdx.y * BLOCKSIZE + 2 * threadIdx.x % 32;

    for (int i1 = 0; i1 < 2; i1++) {
        for (int j1 = 0; j1 < 2; j1++) {
            int i = i0 + i1;
            int j = j0 + j1;
            float tmp = 0;
            for (int k = 0; k < n; k++) {
                tmp += A[n*i+k] * B[n*k+j];
            }
            C[n*i+j] = tmp;
        }
    }
}


void perform_matmul(float *dA, float *dB, float *dC, int n) {

    hipError_t err;
    int threadsPerBlock = 32*32;
    dim3 gridDim(n/64, n/64);
    cuda_matmul<<<gridDim, threadsPerBlock>>>(dA, dB, dC, n);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /*
    float *hC = (float *)malloc(n*n*sizeof(float));
    err = cudaMemcpy(hC, dC, n*n*sizeof(float), cudaMemcpyDeviceToHost);
    if (err != cudaSuccess) {
        fprintf(stderr, "line 56 Failed to copy vector C from device to host (error code %s)!\n", cudaGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    printf("%f\n", hC[0]);
    printf("%f\n", hC[n]);
    */
}

void check_result(float *dC, int n) {
    size_t size = n*n*sizeof(float);
    float *hC = (float *)malloc(size);
    hipError_t err = hipMemcpy(hC, dC, n*n*sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "line 56 Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    float total = 0;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            total += hC[n*i+j];
        }
    }
    printf("Total: %f (should be 1252)\n", total);
    free(hC);
}

int main() {

    int n = 4096;

    float *hA = cpu_matrix(n);
    float *hB = cpu_matrix(n);
    zero_init(hA, n);
    zero_init(hB, n);
    hA[0] = 1;
    hA[1] = 3;
    hB[0] = 6;
    hB[1] = 5; // shouldn't matter
    hB[n] = 7;

    hA[777*n+999] = 20;
    hB[999*n+777] = 61;

    //rand_init(hA, n);
    //rand_init(hB, n);

    float *dA = cuda_matrix(n);
    float *dB = cuda_matrix(n);
    float *dC = cuda_matrix(n);

    cpu_to_cuda(hA, dA, n);
    cpu_to_cuda(hB, dB, n);

    // -----

    perform_matmul(dA, dB, dC, n); // warmup? doesn't seem to actually reduce variance

    struct timespec start, end;
    double elapsed;
    clock_gettime(CLOCK_MONOTONIC, &start);

    int steps = 100;
    for (int step = 0; step < steps; step++) {
        perform_matmul(dA, dB, dC, n);
    }
    hipDeviceSynchronize();

    clock_gettime(CLOCK_MONOTONIC, &end);
    elapsed = (end.tv_sec - start.tv_sec);
    elapsed += (end.tv_nsec - start.tv_nsec) / 1e9;
    float per_step = elapsed/steps;
    printf("Time elapsed: %.6f seconds per step\n", per_step);
    double flops = n*2/per_step;
    flops *= n*n; // split into two steps to avoid casting stuff
    double gflops = flops / 1000000000;
    printf("Gflops: %.3f\n", gflops);

    check_result(dC, n);

    // -----

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    free(hA);
    free(hB);

    hipDeviceReset();
    return 0;
}

