// 3031 Gflops

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<math.h>
#include<stdbool.h>

float *cpu_matrix(int n) {
    size_t size = n * n * sizeof(float);
    float *M = (float *)malloc(size);
    return M;
}

void zero_init(float *M, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            M[n*i+j] = 0;
        }
    }
}

void rand_init(float *M, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            float random_float = (float)rand() / (float)RAND_MAX;
            M[n*i+j] = random_float;
        }
    }
}

float *cuda_matrix(int n) {
    size_t size = n * n * sizeof(float);
    float *dM;
    hipError_t err = hipMalloc((void**)&dM, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    return dM;
}

void cpu_to_cuda(float *hM, float *dM, int n) {
    size_t size = n*n*sizeof(float);
    hipError_t err = hipMemcpy(dM, hM, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "line 23 Failed to copy vector C from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__global__ void cuda_matmul(float *A, float *B, float *C, int n) {
    int i = blockIdx.x * 32 + threadIdx.x / 32;
    int j = blockIdx.y * 32 + threadIdx.x % 32;
    int idx = n * i + j;

    float tmp = 0;
    for (int k = 0; k < n; k++) {
        tmp += A[n*i+k] * B[n*k+j];
    }
    C[idx] = tmp;
}


void perform_matmul(float *dA, float *dB, float *dC, int n) {

    hipError_t err;
    int threadsPerBlock = 1024;
    dim3 gridDim(n/32, n/32);
    cuda_matmul<<<gridDim, threadsPerBlock>>>(dA, dB, dC, n);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /*
    float *hC = (float *)malloc(n*n*sizeof(float));
    err = cudaMemcpy(hC, dC, n*n*sizeof(float), cudaMemcpyDeviceToHost);
    if (err != cudaSuccess) {
        fprintf(stderr, "line 56 Failed to copy vector C from device to host (error code %s)!\n", cudaGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    printf("%f\n", hC[0]);
    printf("%f\n", hC[n]);
    */
}

void check_result(float *dC, int n) {
    size_t size = n*n*sizeof(float);
    float *hC = (float *)malloc(size);
    hipError_t err = hipMemcpy(hC, dC, n*n*sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "line 56 Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    float total = 0;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            total += hC[n*i+j];
        }
    }
    printf("Total: %f (should be 1252)\n", total);
    free(hC);
}

int main() {

    int n = 4096;

    float *hA = cpu_matrix(n);
    float *hB = cpu_matrix(n);
    zero_init(hA, n);
    zero_init(hB, n);
    hA[0] = 1;
    hA[1] = 3;
    hB[0] = 6;
    hB[1] = 5; // shouldn't matter
    hB[n] = 7;

    hA[777*n+999] = 20;
    hB[999*n+777] = 61;

    //rand_init(hA, n);
    //rand_init(hB, n);

    float *dA = cuda_matrix(n);
    float *dB = cuda_matrix(n);
    float *dC = cuda_matrix(n);

    cpu_to_cuda(hA, dA, n);
    cpu_to_cuda(hB, dB, n);

    // -----

    perform_matmul(dA, dB, dC, n); // warmup? doesn't seem to actually reduce variance

    struct timespec start, end;
    double elapsed;
    clock_gettime(CLOCK_MONOTONIC, &start);

    int steps = 100;
    for (int step = 0; step < steps; step++) {
        perform_matmul(dA, dB, dC, n);
    }
    hipDeviceSynchronize();

    clock_gettime(CLOCK_MONOTONIC, &end);
    elapsed = (end.tv_sec - start.tv_sec);
    elapsed += (end.tv_nsec - start.tv_nsec) / 1e9;
    float per_step = elapsed/steps;
    printf("Time elapsed: %.6f seconds per step\n", per_step);
    double flops = n*2/per_step;
    flops *= n*n; // split into two steps to avoid casting stuff
    double gflops = flops / 1000000000;
    printf("Gflops: %.3f\n", gflops);

    check_result(dC, n);

    // -----

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    free(hA);
    free(hB);

    hipDeviceReset();
    return 0;
}

